#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h> // For half-precision data types


void printSummary(int SIZE, double  duration, double durationTransferInit, double durationTransfer, double durationTransfer2) {
    printf("\r============================== MPI CNN Neural Network(Convolution Algorithm) ==============================\n");
    printf("                                                        Developed by Mohsen Gholami - iMohsen02\n");
    printf("                                                        Ferdowsi Uni Of Mashhad(FUM)\n");
    printf("\t\033[96m %-20s \033[91m %-8d \n\033[m", "Matrix Size:", SIZE);
    printf("\t\033[96m %-20s \033[91m %-8d MB\n\033[m", "RAM Usage:", ((SIZE * SIZE) * sizeof(int)) / 1'000'000);
    printf("\n#######################################################################################################\n\n");
    printf("\033[96m %-30s \033[91m %0.0f \033[m %s\n", "Initialize takes", (durationTransferInit), "milliseconds");
    printf("\033[96m %-30s \033[91m %0.0f \033[m %s\n", "Host to Device takes", (durationTransfer ), "milliseconds");
    printf("\033[96m %-30s \033[91m %0.0f \033[m %s\n", "GPU takes", (duration ), "milliseconds");
    printf("\033[96m %-30s \033[91m %0.0f \033[m %s\n", "Device to Host takes", (durationTransfer2 ), "milliseconds");
    printf("\n#######################################################################################################\n\n");
}

void printDeviceProperties(int deviceId = 0) {

    hipDeviceProp_t deviceProp;
    hipError_t error = hipGetDeviceProperties(&deviceProp, 0);
    
    if (error != hipSuccess) {
        std::cerr << "Error: " << hipGetErrorString(error) << "\n";
        return;
    }

        // Print device properties in a table format using printf and ANSI colors
    printf("\r\033[96m%-30s\033[91m%d\033[m\n", "Device ID:", deviceId);
    printf("\033[96m%-30s\033[91m%s\033[m\n", "Name:", deviceProp.name);
    printf("\033[96m%-30s\033[91m%lld bytes\033[m\n", "Total global memory:", deviceProp.totalGlobalMem);
    printf("\033[96m%-30s\033[91m%lld bytes\033[m\n", "Shared memory per block:", deviceProp.sharedMemPerBlock);
    printf("\033[96m%-30s\033[91m%d kHz\033[m\n", "Clock rate:", deviceProp.clockRate);
    printf("\033[96m%-30s\033[91m%d\033[m\n", "Warp size:", deviceProp.warpSize);
    printf("\033[96m%-30s\033[91m%d\033[m\n", "Multiprocessor count:", deviceProp.multiProcessorCount);
    printf("\033[96m%-30s\033[91m%d.%d\033[m\n", "CUDA capability:", deviceProp.major, deviceProp.minor);
    printf("\033[96m%-30s\033[91m%d\033[m\n", "Compute capability:", deviceProp.computeMode);
    printf("\033[96m%-30s\033[91m%d\033[m\n", "Max threads per block:", deviceProp.maxThreadsPerBlock);
    printf("\033[96m%-30s\033[91m(%d, %d, %d)\033[m\n", "Max threads dimension:", deviceProp.maxThreadsDim[0],
           deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    printf("\033[96m%-30s\033[91m(%d, %d, %d)\033[m\n", "Max grid size:", deviceProp.maxGridSize[0],
           deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    }

// CUDA kernel for performing convolution using Tensor Cores (mixed precision)
__global__ void convolutionKernelTensorCore(half *input, half *filter, half *output, int size, int filter_size) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x; // Global row index
    int ty = threadIdx.y + blockIdx.y * blockDim.y; // Global column index

    // Output size calculation
    int output_size = size - filter_size + 1;

    if (tx < output_size && ty < output_size) {
        half sum = __float2half(0.0f); // Initialize sum to 0 in half precision
        for (int i = 0; i < filter_size; ++i) {
            for (int j = 0; j < filter_size; ++j) {
                // Use Tensor Core optimized matrix multiply
                sum = __hadd(sum, __hmul(input[(tx + i) * size + (ty + j)], filter[i * filter_size + j]));
            }
        }
        output[tx * output_size + ty] = sum;
    }
}

// Function to initialize a matrix with random values in half-precision (FP16)
void initializeMatrix(std::vector<half> &matrix, int size) {
    #pragma omp parallel for
    for (int i = 0; i < size * size; ++i) {
        matrix[i] = __float2half(rand() % 50); // Initialize with random values and convert to half
    }
}

// Function to print a matrix
void printMatrix(const std::vector<half> &matrix, int rows, int cols) {
    for (int r = 0; r < rows; ++r) {
        for (int c = 0; c < cols; ++c) {
            std::cout << __half2float(matrix[r * cols + c]) << "\t";
        }
        std::cout << "\n";
    }
}

int main(int argc, char **argv) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <matrix_size>\n";
        return EXIT_FAILURE;
    }

    const int SIZE = std::atoi(argv[1]); // Matrix size
    const int FILTER_SIZE = 2;           // Filter size

    // Use FP16 (half-precision) for both input and filter
    std::vector<half> input(SIZE * SIZE);
    std::vector<half> filter = {__float2half(1), __float2half(0), __float2half(0), __float2half(-1)};
    int output_size = SIZE - FILTER_SIZE + 1;
    std::vector<half> output(output_size * output_size, __float2half(0));

    // Initialize input matrix with random values
    srand(time(0));
        auto startInit = std::chrono::high_resolution_clock::now();

    printf("\rinitializing arrays ...                              ");
    initializeMatrix(input, SIZE);
    auto endInit = std::chrono::high_resolution_clock::now();
    double durationTransferInit = std::chrono::duration_cast<std::chrono::milliseconds>(endInit - startInit).count();


    // Allocate memory on the device
    printf("\rMallocing arrays on device ...                              ");

    half *d_input, *d_filter, *d_output;
    hipMalloc(&d_input, SIZE * SIZE * sizeof(half));
    hipMalloc(&d_filter, FILTER_SIZE * FILTER_SIZE * sizeof(half));
    hipMalloc(&d_output, output_size * output_size * sizeof(half));

    // Copy data to device
    auto startC = std::chrono::high_resolution_clock::now();
    printf("\rCopying arrays to device ...                              ");
    hipMemcpy(d_input, input.data(), SIZE * SIZE * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_filter, filter.data(), FILTER_SIZE * FILTER_SIZE * sizeof(half), hipMemcpyHostToDevice);
    auto endC = std::chrono::high_resolution_clock::now();
    double durationTransfer = std::chrono::duration_cast<std::chrono::milliseconds>(endC - startC).count();

    // Launch the kernel
    dim3 blockDim(16, 16); // Block of 16x16 threads
    dim3 gridDim((output_size + 15) / 16, (output_size + 15) / 16);
    auto start = std::chrono::high_resolution_clock::now();
    convolutionKernelTensorCore<<<gridDim, blockDim>>>(d_input, d_filter, d_output, SIZE, FILTER_SIZE);

    printf("\rRunning Kernel ...                              ");
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    // Copy results back to host
    auto startC2 = std::chrono::high_resolution_clock::now();
    printf("\rCoping output to host ...                              ");
    hipMemcpy(output.data(), d_output, output_size * output_size * sizeof(half), hipMemcpyDeviceToHost);
    auto endC2 = std::chrono::high_resolution_clock::now();
    double durationTransfer2 = std::chrono::duration_cast<std::chrono::milliseconds>(endC2 - startC2).count();
    // // Print results
    // std::cout << "\nOutput Matrix:\n";
    // printMatrix(output, output_size, output_size);

    // Display execution time
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    printDeviceProperties();
    printSummary(SIZE, duration,durationTransferInit,durationTransfer,durationTransfer2);
    // Free device memory
    hipFree(d_input);
    hipFree(d_filter);
    hipFree(d_output);

    return EXIT_SUCCESS;
}
