#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h> // Removed, not used for int8 anymore
#include <stdint.h>    // For int8_t type

void printSummary(int SIZE, double duration, double durationTransferInit, double durationTransfer, double durationTransfer2)
{
    printf("\r============================== MPI CNN Neural Network(Convolution Algorithm) ==============================\n");
    printf("                                                        Developed by Mohsen Gholami - iMohsen02\n");
    printf("                                                        Ferdowsi Uni Of Mashhad(FUM)\n");
    printf("\t\033[96m %-20s \033[91m %-8d \n\033[m", "Matrix Size:", SIZE);
    printf("\t\033[96m %-20s \033[91m %-8d MB\n\033[m", "RAM Usage:", ((SIZE * SIZE) * sizeof(int)) / 1'000'000);
    printf("\n#######################################################################################################\n\n");
    printf("\033[96m %-30s \033[91m %0.0f \033[m %s\n", "Initialize takes", (durationTransferInit), "milliseconds");
    printf("\033[96m %-30s \033[91m %0.0f \033[m %s\n", "Host to Device takes", (durationTransfer), "milliseconds");
    printf("\033[96m %-30s \033[91m %0.0f \033[m %s\n", "GPU takes", (duration), "milliseconds");
    printf("\033[96m %-30s \033[91m %0.0f \033[m %s\n", "Device to Host takes", (durationTransfer2), "milliseconds");
    printf("\n#######################################################################################################\n\n");
}

void printDeviceProperties(int deviceId = 0)
{

    hipDeviceProp_t deviceProp;
    hipError_t error = hipGetDeviceProperties(&deviceProp, 0);

    if (error != hipSuccess)
    {
        std::cerr << "Error: " << hipGetErrorString(error) << "\n";
        return;
    }

    // Print device properties in a table format using printf and ANSI colors
    printf("\r\033[96m%-30s\033[91m%d\033[m\n", "Device ID:", deviceId);
    printf("\033[96m%-30s\033[91m%s\033[m\n", "Name:", deviceProp.name);
    printf("\033[96m%-30s\033[91m%lld bytes\033[m\n", "Total global memory:", deviceProp.totalGlobalMem);
    printf("\033[96m%-30s\033[91m%lld bytes\033[m\n", "Shared memory per block:", deviceProp.sharedMemPerBlock);
    printf("\033[96m%-30s\033[91m%d kHz\033[m\n", "Clock rate:", deviceProp.clockRate);
    printf("\033[96m%-30s\033[91m%d\033[m\n", "Warp size:", deviceProp.warpSize);
    printf("\033[96m%-30s\033[91m%d\033[m\n", "Multiprocessor count:", deviceProp.multiProcessorCount);
    printf("\033[96m%-30s\033[91m%d.%d\033[m\n", "CUDA capability:", deviceProp.major, deviceProp.minor);
    printf("\033[96m%-30s\033[91m%d\033[m\n", "Compute capability:", deviceProp.computeMode);
    printf("\033[96m%-30s\033[91m%d\033[m\n", "Max threads per block:", deviceProp.maxThreadsPerBlock);
    printf("\033[96m%-30s\033[91m(%d, %d, %d)\033[m\n", "Max threads dimension:", deviceProp.maxThreadsDim[0],
           deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    printf("\033[96m%-30s\033[91m(%d, %d, %d)\033[m\n", "Max grid size:", deviceProp.maxGridSize[0],
           deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
}

// CUDA kernel for performing convolution using Tensor Cores (int8)
__global__ void convolutionKernelTensorCoreInt8(int8_t *input, int8_t *filter, int8_t *output, int size, int filter_size)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    int output_size = size - filter_size + 1;

    if (tx < output_size && ty < output_size)
    {
        int32_t sum = 0;
        for (int i = 0; i < filter_size; ++i)
        {
            for (int j = 0; j < filter_size; ++j)
            {

                sum += input[(tx + i) * size + (ty + j)] * filter[i * filter_size + j];
            }
        }
        output[tx * output_size + ty] = static_cast<int8_t>(sum);
    }
}

// Function to initialize a matrix with random values in int8
void initializeMatrix(std::vector<int8_t> &matrix, int size)
{
#pragma omp parallel for
    for (int i = 0; i < size * size; ++i)
    {
        matrix[i] = rand() % 50; // Initialize with random values in int8 range
    }
}

// Function to print a matrix
void printMatrix(const std::vector<int8_t> &matrix, int rows, int cols)
{
    for (int r = 0; r < rows; ++r)
    {
        for (int c = 0; c < cols; ++c)
        {
            std::cout << static_cast<int>(matrix[r * cols + c]) << "\t"; // Print as int
        }
        std::cout << "\n";
    }
}

int main(int argc, char **argv)
{
    if (argc < 2)
    {
        std::cerr << "Usage: " << argv[0] << " <matrix_size>\n";
        return EXIT_FAILURE;
    }

    const int SIZE = std::atoi(argv[1]); // Matrix size
    const int FILTER_SIZE = 2;           // Filter size

    // Use int8_t for both input and filter
    std::vector<int8_t> input(SIZE * SIZE);
    std::vector<int8_t> filter = {1, 0, 0, -1}; // 2x2 filter with int8_t values
    int output_size = SIZE - FILTER_SIZE + 1;
    std::vector<int8_t> output(output_size * output_size, 0);

    // Initialize input matrix with random values
    srand(time(0));
    auto startInit = std::chrono::high_resolution_clock::now();

    printf("\rinitializing arrays ...                              ");
    initializeMatrix(input, SIZE);
    auto endInit = std::chrono::high_resolution_clock::now();
    double durationTransferInit = std::chrono::duration_cast<std::chrono::milliseconds>(endInit - startInit).count();

    // Allocate memory on the device
    printf("\rMallocing arrays on device ...                              ");

    int8_t *d_input, *d_filter, *d_output;
    hipMalloc(&d_input, SIZE * SIZE * sizeof(int8_t));
    hipMalloc(&d_filter, FILTER_SIZE * FILTER_SIZE * sizeof(int8_t));
    hipMalloc(&d_output, output_size * output_size * sizeof(int8_t));

    // Copy data to device
    auto startC = std::chrono::high_resolution_clock::now();
    printf("\rCopying arrays to device ...                              ");
    hipMemcpy(d_input, input.data(), SIZE * SIZE * sizeof(int8_t), hipMemcpyHostToDevice);
    hipMemcpy(d_filter, filter.data(), FILTER_SIZE * FILTER_SIZE * sizeof(int8_t), hipMemcpyHostToDevice);
    auto endC = std::chrono::high_resolution_clock::now();
    double durationTransfer = std::chrono::duration_cast<std::chrono::milliseconds>(endC - startC).count();

    // Launch the kernel
    dim3 blockDim(16, 16); // Block of 16x16 threads
    dim3 gridDim((output_size + 15) / 16, (output_size + 15) / 16);
    auto start = std::chrono::high_resolution_clock::now();
    convolutionKernelTensorCoreInt8<<<gridDim, blockDim>>>(d_input, d_filter, d_output, SIZE, FILTER_SIZE);

    printf("\rRunning Kernel ...                              ");
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    // Copy results back to host
    auto startC2 = std::chrono::high_resolution_clock::now();
    printf("\rCoping output to host ...                              ");
    hipMemcpy(output.data(), d_output, output_size * output_size * sizeof(int8_t), hipMemcpyDeviceToHost);
    auto endC2 = std::chrono::high_resolution_clock::now();
    double durationTransfer2 = std::chrono::duration_cast<std::chrono::milliseconds>(endC2 - startC2).count();

    // Display execution time
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    printDeviceProperties();
    printSummary(SIZE, duration, durationTransferInit, durationTransfer, durationTransfer2);

    // Free device memory
    hipFree(d_input);
    hipFree(d_filter);
    hipFree(d_output);

    return EXIT_SUCCESS;
}
